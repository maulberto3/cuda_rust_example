// elementwise_product.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void elementwiseProduct(float* A, float* B, float* C, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] * B[idx];
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(float);

    // Allocate memory on the host
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize arrays on the host
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy input data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    elementwiseProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy output data from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result
    printf("Element-wise product:\n");
    for (int i = 0; i < N; ++i) {
        printf("%f * %f = %f\n", h_A[i], h_B[i], h_C[i]);
    }

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
